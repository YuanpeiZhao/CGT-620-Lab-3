#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include ""
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "device_atomic_functions.h"

#include <time.h>
#include <windows.h>

#define N 2048

#define M 1024
#define T 100
int a[N], b[N], c[N];

using namespace std;

__global__ void add(int* a, int* b, int* c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

void FillArrays() {

    for (int i = 0; i < N; i++) {
        srand(i);
        a[i] = rand() % 1000000;
        b[i] = rand() & 1000000;
    }
    return;
}

float add_serial() {

    float time;
    LARGE_INTEGER t1, t2, tc;
    QueryPerformanceFrequency(&tc);
    QueryPerformanceCounter(&t1);

    FillArrays();

    QueryPerformanceCounter(&t2);
    time = (float)(t2.QuadPart - t1.QuadPart) / (double)tc.QuadPart * 1000;
  
    for (int i = 0; i < N; i++)
        c[i] = a[i] + b[i];

    return time;
}

float add_parallel_explicit() {

    int* d_a, * d_b, * d_c;

    hipEvent_t startT, stopT;
    float time;
    hipEventCreate(&startT);
    hipEventCreate(&stopT);
    hipEventRecord(startT, 0);

    hipMalloc((void**)&d_a, sizeof(int) * N);
    hipMalloc((void**)&d_b, sizeof(int) * N);
    hipMalloc((void**)&d_c, sizeof(int) * N);

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add << <(N + M - 1) / M, M >> > (d_a, d_b, d_c);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stopT, 0);
    hipEventSynchronize(stopT);
    hipEventElapsedTime(&time, startT, stopT);
    hipEventDestroy(startT);
    hipEventDestroy(stopT);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


    
    return time;
}

float add_parallel_unified() {

    int* aa, * bb, * cc;

    hipEvent_t startT, stopT;
    float time, time1;
    

    LARGE_INTEGER t1, t2, tc;
    QueryPerformanceFrequency(&tc);
    QueryPerformanceCounter(&t1);

    hipMallocManaged(&aa, N * sizeof(int));
    hipMallocManaged(&bb, N * sizeof(int));
    hipMallocManaged(&cc, N * sizeof(int));

    QueryPerformanceCounter(&t2);
    time1 = (float)(t2.QuadPart - t1.QuadPart) / (double)tc.QuadPart * 1000;
    
    for (int i = 0; i < N; i++) {
        srand(i);
        aa[i] = rand() % 1000000;
        bb[i] = rand() & 1000000;
    }

    hipEventCreate(&startT);
    hipEventCreate(&stopT);
    hipEventRecord(startT);

    add << <(N + M - 1) / M, M >> > (aa, bb, cc);

    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "error! : %s\n", hipGetErrorString(cudaErr));
    }

    hipEventRecord(stopT);
    hipEventSynchronize(stopT);
    hipEventElapsedTime(&time, startT, stopT);
    hipEventDestroy(startT);
    hipEventDestroy(stopT);

    hipFree(aa);
    hipFree(bb);
    hipFree(cc);

    return time + time1;
}



int main() {

    float t1 = 0, t2 = 0, t3 = 0;

    for (int i = 0; i < T; i++) {
        //t1 += add_serial() / T;
        if (i == 1) {
            t2 = 0;
            t3 = 0;
        }
        t2 += add_parallel_explicit() / (T-1);
        t3 += add_parallel_unified() / (T-1);
        cout << "iteration: " << i << endl;
    }



    //cout << "serial time:\t" << t1 << endl;
    cout << "parallel explicit time:\t" << t2 << endl;
    cout << "parallel unified time:\t" << t3 << endl;

    return 0;
}